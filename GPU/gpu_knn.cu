#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <helper_functions.h>
#include <stdlib.h>
#include <stdio.h>
#include "../configuration.h"
#include "gpu_knn.h"

const int blocksize = 16;
__global__ void gpu_distance(int* data, float* distance, int* point, int n, int dim) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i >= n)
		return;

	float d = 0;

        for(int j = 0; j<dim; j++)
                d += abs(data[i*dim + j] - point[j]);

	distance[i] = d;	
}

int gpu_knn(int * cdata_c, int * data_c, int * point_c, int nclass) {
	
	int datasize = N*DIM*sizeof(int);
	int nblock = N / blocksize, nthread = blocksize;
	
	float *distance = new float[N]; 

	int *d_data;
	int *d_point;	
	float *d_distance;

	checkCudaErrors(hipMalloc((void**)&d_data, datasize));	
	checkCudaErrors(hipMalloc((void**)&d_distance, N*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_point, DIM*sizeof(int)));

	checkCudaErrors(hipMemcpy(d_data, data_c, datasize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_point, point_c, DIM*sizeof(int), hipMemcpyHostToDevice));

	gpu_distance<<<nblock,nthread>>>(d_data,d_distance,d_point,N,DIM);

	checkCudaErrors(hipMemcpy(distance, d_distance, N*sizeof(float), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_distance)); 
 	checkCudaErrors(hipFree(d_data)); 
 	checkCudaErrors(hipFree(d_point)); 
	
 	printf("Memory 6 \n");
	return -1;
}



