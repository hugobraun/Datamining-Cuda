#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <helper_functions.h>
#include <stdlib.h>
#include <stdio.h>
#include "../configuration.h"


#include <stdio.h>

#include <stdlib.h>

#include "gpu_sort.h"


/*
 * Naive sort
 * used if the quicksort uses too many levels
 */
__device__ void naivesort(int *data, int left, int right )
{
  for(int i = left ; i <= right ; ++i)
  {
    int min = data[i];
    int imin= i;

    for(int j = i+1 ; j <= right ; ++j)
    {
      int vj = data[j];
      if(vj < min)
      {
        imin = j;
        min = vj;
      }
    }

    if(i != imin)
    {
      data[imin] = data[i];
      data[i] = min;
    }
  }
}

/*
 * The idea behind that approach is that sorting an already sorted list is really fast
 * We do not need to warn the thread that the portion is already sorted
 *
 *
 * TODO : max_levels
 */
__global__ void kernel_quicksort(int* values, int n) {
 #define MAX_LEVELS	1000

	int pivot, L, R;
	int idx =  threadIdx.x + blockIdx.x * blockDim.x;
	int start[MAX_LEVELS];
	int end[MAX_LEVELS];

	start[idx] = idx;
	end[idx] = n - 1;
	while (idx >= 0) {
		L = start[idx];
		R = end[idx];
		if (L < R) {
			pivot = values[L];
			while (L < R) {
				while (values[R] >= pivot && L < R)
					R--;
				if(L < R)
					values[L++] = values[R];
				while (values[L] < pivot && L < R)
					L++;
				if (L < R)
					values[R--] = values[L];
			}
			values[L] = pivot;

			start[idx + 1] = L + 1;
			end[idx + 1] = end[idx];
			end[idx++] = L;


			if (end[idx] - start[idx] > end[idx - 1] - start[idx - 1]) {
        	                int tmp = start[idx];
                	        start[idx] = start[idx - 1];
                        	start[idx - 1] = tmp;

        	                tmp = end[idx];
                	        end[idx] = end[idx - 1];
                        	end[idx - 1] = tmp;
	        }

		}
		else
			idx--;
	}
}

void gpu_quicksort(int * data, int n) {

	int datasize = n*sizeof(int);
	int *d_data;
	checkCudaErrors(hipMalloc((void**)&d_data, datasize));
	checkCudaErrors(hipMemcpy(d_data, data, datasize, hipMemcpyHostToDevice));

	kernel_quicksort<<<1,1,128>>>(d_data,n);

	checkCudaErrors(hipMemcpy(data, d_data, datasize, hipMemcpyDeviceToHost));

 	checkCudaErrors(hipFree(d_data));
}

void gpu_quicksort_benchmark(int * data, int n, int ntests) {
	printf("------------------------------------------\n");
	printf("Starting benchmark for gpu quicksort (ntests = %i, array size = %i)\n", ntests, n);
	printf("------------------------------------------\n");

	StopWatchInterface *timer = 0;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	for(int i =0; i < ntests; i++)
		gpu_quicksort(data, n);

	sdkStopTimer(&timer);

	printf("------------------------------------------\n");
	printf("Sorting of array size %i done. Processing time on GPU per job: %f (ms)\n",n,sdkGetTimerValue(&timer)/ntests);
	printf("------------------------------------------\n");

	printf("\nTesting results...\n");
	for (int x = 0; x < n - 1; x++) {
		if (data[x] > data[x + 1]) {
				printf("Sorting failed.\n");
				break;
		}
		else
			if (x == n - 2)
					printf("SORTING SUCCESSFUL\n");
	}


}
